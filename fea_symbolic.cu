#include "hip/hip_runtime.h"
//ginac + nvcc
#include <iostream>
#include <ctime>
#include <vector>
#include <ginac/ginac.h>
#include <chrono>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <sstream>      // std::ostringstream

#include "hip/hip_runtime.h"
#include ""

using namespace GiNaC;
using namespace std;

/////////////////////////////////////////////////////////////////////

static ex sfR_eval (const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3);
static ex sfR_deriv(const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3, unsigned diff_param);
static ex sfS_eval (const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3);
static ex sfS_deriv(const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3, unsigned diff_param);

DECLARE_FUNCTION_8P(sfR)
REGISTER_FUNCTION(sfR, eval_func(sfR_eval).
                             derivative_func(sfR_deriv).
                             latex_name("r"));
DECLARE_FUNCTION_8P(sfS)
REGISTER_FUNCTION(sfS, eval_func(sfS_eval).
                             derivative_func(sfS_deriv).
                             latex_name("s"));

ex sfR_eval (const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3) 
{
    return sfR(x,y,x1,x2,x3,y1,y2,y3).hold();
}
//    r_x = (y2-y3)/jac;
//    r_y = (x3-x2)/jac;
ex sfR_deriv(const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3, unsigned diff_param) 
{
  symbol r("r"), s("s");
  ex fx = x1*r + x2*s + x3*(1-r-s);
  ex fy = y1*r + y2*s + y3*(1-r-s);
  ex jac = fx.diff(r)*fy.diff(s) - fy.diff(r)*fx.diff(s);
  if(diff_param == 0)
      return (y2-y3)/jac;
  else if(diff_param == 1)
    return (x3-x2)/jac;
  else
    return 0;
}
ex sfS_eval (const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3) 
{
    return sfS(x,y,x1,x2,x3,y1,y2,y3).hold();
}
//    s_x = (y3-y1)/jac;
//    s_y = (x1-x3)/jac;
ex sfS_deriv(const ex &x, const ex &y, const ex &x1, const ex &x2, const ex &x3, const ex &y1, const ex &y2, const ex &y3, unsigned diff_param) 
{
  symbol r("r"), s("s");
  ex fx = x1*r + x2*s + x3*(1-r-s);
  ex fy = y1*r + y2*s + y3*(1-r-s);
  ex jac = fx.diff(r)*fy.diff(s) - fy.diff(r)*fx.diff(s);
  if(diff_param == 0)
      return (y3-y1)/jac;
  else if(diff_param == 1)
    return (x1-x3)/jac;
  else
    return 0;
}

/////////////////////////////////////////////////////
lst grad(ex &f, symbol &x, symbol &y) 
{
  return lst(f.diff(x), f.diff(y));
}

ex dot(lst l, lst r) 
{
  ex ret = 0;
  for (size_t i = 0; i < l.nops(); ++i)
       ret += l[i]*r[i];
  return ret;
}

//////////////////////////////////////////////////////
class Node 
{
public:
  double x, y, z;
  int flag; //boundary flag
  int index; //global index of node
};

class Element 
{
public:
  vector<Node*> nodes;
};

class Mesh 
{
public:
  vector<Node*> nodes;
  vector<Element*> elements;
  void printMesh()
  {
    cout << "number of nodes = " << nodes.size() << endl;
    for(int i=0; i<nodes.size(); i++)
    {
      Node *node = nodes[i];
      cout << node->index << " " <<node->x << " " << node->y << " " << node->flag << endl;
    }
    cout << "number of elements = " <<elements.size() << endl;
    for(int i=0; i<elements.size(); i++)
    {
      Element *e = elements[i];
      cout << e->nodes[0]->index << " " << e->nodes[1]->index << " " << e->nodes[2]->index << endl;
    }
  }
};

class RectangleMesh : public Mesh 
{
public:
  double x0,x1,y0,y1;
  int nRow, nCol;
  RectangleMesh(double x0, double x1, double y0, double y1, int nRow, int nCol) 
  {
    this->x0 = x0;
    this->x1 = x1;
    this->y0 = y0;
    this->y1 = y1;
    this->nRow = nRow;
    this->nCol = nCol;
    generate();
  }

  void generate()
  {
    double stepx = (x1-x0)/nCol;
    double stepy = (y1-y0)/nRow;
    //generate nodes
    for(int i=0; i<=nRow; i++)
    {
      double y = y0+i*stepy;
      for(int j=0; j<=nCol; j++)
      {
        double x = x0+j*stepx;
        Node *node = new Node();
        node->x = x;
        node->y = y;
        if(i==0 || i==nRow || j==0 || j==nCol)
          node->flag = 1; //on the bounday
        else
          node->flag = 0;
        node->index = i*(nCol+1) + j;
        nodes.push_back(node);
      }
    }
    //generate elements
    for(int i=0; i<nRow; i++)
    {
      for(int j=0; j<nCol; j++)
      {
        Element *e = new Element();
        int n1 = i*(nCol+1) + j;
        int n2 = n1 + 1;
        int n3 = (i+1)*(nCol+1) + j;
        e->nodes.push_back(nodes[n1]);
        e->nodes.push_back(nodes[n2]);
        e->nodes.push_back(nodes[n3]);
        elements.push_back(e);

        e = new Element();
        n1 = i*(nCol+1) + j + 1;
        n2 = (i+1)*(nCol+1) + j+ 1;
        n3 = n2 - 1;
        e->nodes.push_back(nodes[n1]);
        e->nodes.push_back(nodes[n2]);
        e->nodes.push_back(nodes[n3]);
        elements.push_back(e);
      }
    }
  }
};

class UnitSquareMesh : public RectangleMesh
{
public:
  UnitSquareMesh(int nRow, int nCol) :
    RectangleMesh(0.0,1.0,0.0,1.0,nRow,nCol) {}
};

class FunctionSpace 
{
public:
  Mesh mesh;
  lst coords;     //x,y
  lst refCoords;  //r,s
  lst nodeCoords; //x1,x2,x3,y1,y2,y3
  lst sfRS;       //r(x,y), s(x,y)
  lst trans;
  FunctionSpace()
  {

  }
  FunctionSpace(Mesh &mesh, lst coords, string type, int order) 
  {
    this->mesh = mesh;

    this->coords = coords;
    
    ex x = coords[0];
    ex y = coords[1];

    symbol r("r"), s("s");
    refCoords = lst(r, s);

    symbol x1("x1"), x2("x2"), x3("x3");
    symbol y1("y1"), y2("y2"), y3("y3");
    nodeCoords = x1, x2, x3, y1, y2, y3;

    ex sfr = sfR(x,y,x1,x2,x3,y1,y2,y3);
    ex sfs = sfS(x,y,x1,x2,x3,y1,y2,y3);    
    sfRS = lst(sfr, sfs);

    ex fx = x1*r + x2*s + x3*(1-r-s);
    ex fy = y1*r + y2*s + y3*(1-r-s);
    trans = lst(fx, fy);
  }

  lst getShapeFunctions() 
  {
    ex sfr = sfRS[0];
    ex sfs = sfRS[1];
    return lst(sfr, sfs, 1-sfr-sfs);
  }

  int getDOFNum()
  {
    return 3;
  }

  lst getTransform() 
  {
    return trans;
  }

  ex getJac() 
  {
    symbol r = ex_to<symbol>(refCoords[0]);
    symbol s = ex_to<symbol>(refCoords[1]);
    lst trans = getTransform();
    ex fx = trans[0];
    ex fy = trans[1];
    return fx.diff(r)*fy.diff(s) - fy.diff(r)*fx.diff(s);
  }
};

///////////////////////////////////////////////////////
class WeakForm 
{
public:
  FunctionSpace funcSpace;
  ex lhs[3][3];
  ex rhs[3];

  WeakForm(FunctionSpace &funcSpace) 
  {
    this->funcSpace = funcSpace;
  }

  void build(std::function<ex(ex,ex)> _lhs, std::function<ex(ex)> _rhs) 
  {
    lst sfuncs = funcSpace.getShapeFunctions();
    ex sfr = sfuncs[0];
    ex sfs = sfuncs[1];

    int nDOFs = funcSpace.getDOFNum();
    ex x = funcSpace.coords[0];
    ex y = funcSpace.coords[1];
    lst trans = funcSpace.getTransform();
    ex fx = trans[0];
    ex fy = trans[1];

    ex jac = funcSpace.getJac();

    lst argsOrder;
    ex r = funcSpace.refCoords[0];
    ex s = funcSpace.refCoords[1];
    ex x1 = funcSpace.nodeCoords[0];
    ex x2 = funcSpace.nodeCoords[1];
    ex x3 = funcSpace.nodeCoords[2];
    ex y1 = funcSpace.nodeCoords[3];
    ex y2 = funcSpace.nodeCoords[4];
    ex y3 = funcSpace.nodeCoords[5];    
    argsOrder = x1,x2,x3,y1,y2,y3,r,s;
    std::ostringstream oss;
    for(int j=0; j<nDOFs; j++) 
    {
      for(int i=0; i<nDOFs; i++) 
      {
        lhs[j][i] =  _lhs(sfuncs[j], sfuncs[i]).subs(lst(sfr==r, sfs==s)).subs(lst(x==fx, y==fy))*jac;
        //cout<<csrc_float<<lhs[j][i]<<endl;
        oss.str(""); oss.clear();
        oss<<csrc_float<<lhs[j][i]<<endl;
        printf("%s", oss.str().c_str());
      }
      rhs[j] = _rhs(sfuncs[j]).subs(lst(sfr==r, sfs==s)).subs(lst(x==fx, y==fy))*jac;
      //cout<<csrc_float<<rhs[j]<<endl;
      oss.str(""); oss.clear();
      oss<<csrc_float<<rhs[j]<<endl;
      printf("%s", oss.str().c_str());
    }
  }
/*
-( 1.0/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)*std::pow( x2-x3,2.0)+std::pow( y2-y3,2.0)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
( ( x1-x3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)*( x2-x3)+( y1-y3)*( y2-y3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( ( y2-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))
2.0*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*r*( std::pow( y1*r+s*y2-( s+r-1.0)*y3,2.0)+std::pow( s*x2+x1*r-( s+r-1.0)*x3,2.0)-18.0)
( ( x1-x3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)*( x2-x3)+( y1-y3)*( y2-y3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
-( std::pow( x1-x3,2.0)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)+std::pow( y1-y3,2.0)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
( ( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))*( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
2.0*s*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( std::pow( y1*r+s*y2-( s+r-1.0)*y3,2.0)+std::pow( s*x2+x1*r-( s+r-1.0)*x3,2.0)-18.0)
( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( ( y2-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))
( ( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))*( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
-( std::pow( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3),2.0)+std::pow( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))
-2.0*( s+r-1.0)*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( std::pow( y1*r+s*y2-( s+r-1.0)*y3,2.0)+std::pow( s*x2+x1*r-( s+r-1.0)*x3,2.0)-18.0)

( 1.0/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*pow( y2-y3,2.0)+1.0/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*pow( x2-x3,2.0))*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))
-( ( y1-y3)/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*( y2-y3)+1.0/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*( x1-x3)*( x2-x3))*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))
( ( 1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3)-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3))/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3)+1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( y2-y3))*( y2-y3))*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))
-2.0*( pow( y2*s+y1*r-( r+s-1.0)*y3,2.0)+pow( x2*s-( r+s-1.0)*x3+x1*r,2.0)-18.0)*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*r
-( ( y1-y3)/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*( y2-y3)+1.0/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*( x1-x3)*( x2-x3))*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))
( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( pow( y1-y3,2.0)/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)+1.0/pow( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3),2.0)*pow( x1-x3,2.0))
-( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( y2-y3))+( 1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3)-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3))/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3))
-2.0*( pow( y2*s+y1*r-( r+s-1.0)*y3,2.0)+pow( x2*s-( r+s-1.0)*x3+x1*r,2.0)-18.0)*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*s
( ( 1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3)-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3))/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3)+1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( y2-y3))*( y2-y3))*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))
-( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( y2-y3))+( 1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3)-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3))/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3))
( pow( 1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x1-x3)-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( x2-x3),2.0)+pow( ( y1-y3)/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))-1.0/( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))*( y2-y3),2.0))*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))
2.0*( pow( y2*s+y1*r-( r+s-1.0)*y3,2.0)+pow( x2*s-( r+s-1.0)*x3+x1*r,2.0)-18.0)*( r+s-1.0)*( ( x1-x3)*( y2-y3)-( y1-y3)*( x2-x3))


*/
  void assemble() {

  }

};

//////////////////////////////////////////////////////////////

#define MESH_W 4
#define MESH_H 4

#define M (MESH_W+1)*(MESH_H+1) //size of matrix A M by N
#define N (MESH_W+1)*(MESH_H+1)
#define NE 2*MESH_W*MESH_H //number of elements

#define BLOCK_X 7 // number of integration points
#define BLOCK_Y 9 // number of expressions
#define BLOCK_Z ((int)(32*32)/(BLOCK_X*BLOCK_Y)) //number of elements in a block
#define NDOF 3 //number of DOFs
#define NNODE 3 //number of nodes

__constant__ float triW[7] = { 0.06296959f, 0.06619708f, 0.06296959f, 0.06619708f, 0.06296959f, 0.06619708f, 0.11250000f };
__constant__ float triR[7] = { 0.10128651f, 0.47014206f, 0.79742699f, 0.47014206f, 0.10128651f, 0.05971587f, 0.33333333f };
__constant__ float triS[7] = { 0.10128651f, 0.05971587f, 0.10128651f, 0.47014206f, 0.79742699f, 0.47014206f, 0.33333333f };
__constant__ float triT[7] = { 0.79742698f, 0.47014207f, 0.1012865f,  0.05971588f, 0.1012865f,  0.47014207f, 0.33333334f };


//This function should be generated from the symbol expressions of the integrand
__device__ float integrand(int funIdx, float *params)
{
  float x1 = params[0];
  float x2 = params[1];
  float x3 = params[2];
  float y1 = params[3];
  float y2 = params[4];
  float y3 = params[5];
  float r = params[6];
  float s = params[7];
  float t = params[8];
  //printf("%d %f %f %f %f %f %f %f %f %f\n", funIdx, x1,x2,x3,y1,y2,y3,r,s,t);
  if(funIdx == 0)
    return -( 1.0/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)*std::pow( x2-x3,2.0)+std::pow( y2-y3,2.0)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  if(funIdx == 1)
    return ( ( x1-x3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)*( x2-x3)+( y1-y3)*( y2-y3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  if(funIdx == 2)
    return ( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( ( y2-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3));
  if(funIdx == 3)
    return ( ( x1-x3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)*( x2-x3)+( y1-y3)*( y2-y3)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  if(funIdx == 4)
    return -( std::pow( x1-x3,2.0)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0)+std::pow( y1-y3,2.0)/std::pow( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  if(funIdx == 5)
    return ( ( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))*( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  if(funIdx == 6)
    return ( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( ( y2-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3));
  if(funIdx == 7)
    return ( ( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3))*( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)*( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  if(funIdx == 8)
    return -( std::pow( ( x1-x3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-1.0/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))*( x2-x3),2.0)+std::pow( ( y2-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3))-( y1-y3)/( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3)),2.0))*( ( y1-y3)*( x2-x3)-( y2-y3)*( x1-x3));
  return 0.0f;
}


//Version 3: use shared memory
__global__ void fea_kernel(float* A, 
    float *X, float *Y, // (x,y) of each element for all the element
    int *gIdx // node index of each element for all the element
  )
{
  __shared__ float sX[BLOCK_Z*NNODE];    //shared memory of copy of X in the current block
  __shared__ float sY[BLOCK_Z*NNODE];    //shared memory of copy of Y in the current block
  __shared__ float sGIdx[BLOCK_Z*NNODE]; //shared memory of copy of gIdx in the current block

  int gEleIdx = BLOCK_Z*blockIdx.x + threadIdx.z; //global element index 
  int sEleIdx = NNODE*threadIdx.z;                //global element index in shared memory arrays: sX,sY,sGIdx

  // copy from global memory to shared memory for X, Y and gIdx
  if(threadIdx.x==0 && threadIdx.y==0)
  {
#pragma unroll
    for(int i=0; i<NNODE; i++)
      sX[sEleIdx+i]=X[NNODE*gEleIdx+i];

#pragma unroll
    for(int i=0; i<NNODE; i++)
      sY[sEleIdx+i]=Y[NNODE*gEleIdx+i];

#pragma unroll
    for(int i=0; i<NNODE; i++)
      sGIdx[sEleIdx+i]=gIdx[NNODE*gEleIdx+i];
  }
  __syncthreads();

  //local matrix row and column index
  //threadIdx.y = 0,1,2,3,4,5,6,7,8 (BLOCK_Y)
  int li = threadIdx.y / NDOF;
  int lj = threadIdx.y % NDOF;
  __shared__ float localFlatMatrix[BLOCK_Y*BLOCK_Z]; //array for the local flat matrices of all the elememnts in the current block
  int lfmIdx = threadIdx.z*BLOCK_Y + threadIdx.y; //local flat matrix index of the integrand of threadIdx.y
  float params[3*NNODE]; //parameters array of integrand

  //compute local matrix
  if(gEleIdx < NE)
  {
#pragma unroll
    for(int i=0; i<NNODE; i++)
      params[i] = sX[sEleIdx+i];

#pragma unroll
    for(int i=0; i<NNODE; i++)
      params[NNODE+i] = sY[sEleIdx+i];

    params[2*NNODE+0] = triR[threadIdx.x];
    params[2*NNODE+1] = triS[threadIdx.x];
    params[2*NNODE+2] = triT[threadIdx.x]; //triT[threadIdx.x]=1.0-triR[threadIdx.x]-triS[threadIdx.x];

    atomicAdd( &localFlatMatrix[lfmIdx], triW[threadIdx.x]*integrand(threadIdx.y, params) );
  }
  __syncthreads();

  //write to gobal matrix A
  if(gEleIdx < NE)
  {
    if(threadIdx.x == 0)
    {
      //global matrix row and column index
      int gi  = sGIdx[sEleIdx + li];
      int gj  = sGIdx[sEleIdx + lj];
      atomicAdd( &A[N*gj + gi], localFlatMatrix[lfmIdx] );
    }
  }
}

hipError_t assembleWithCuda()
{
    dim3 dim_block;
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float elapsed = 0;

    dim_block.x = BLOCK_X;
    dim_block.y = BLOCK_Y;
    dim_block.z = BLOCK_Z;

    printf("block_x:%d, block_y:%d, block_z:%d\n", dim_block.x, dim_block.y, dim_block.z);

    hipSetDevice(0);

    RectangleMesh mesh(-3.0, 3.0, -3.0, 3.0, MESH_W, MESH_H);
    mesh.printMesh();

    symbol x("x"), y("y");
    ex f = -2*(x*x + y*y) + 36; //Right hand side(RHS)

    FunctionSpace fs = FunctionSpace(mesh, lst(x, y), "Lagrange", 1);

    WeakForm wf(fs);
    wf.build(
      [&](ex u, ex v) { return dot(grad(u,x,y), grad(v,x,y)); },
      [&](ex v) { return f*v; }
    );

    float *A  = (float*)malloc( M*N*sizeof(float) );
    float *X  = (float*)malloc( NE*NNODE*sizeof(float) );
    float *Y  = (float*)malloc( NE*NNODE*sizeof(float) );
    int *gIdx = (int*)malloc( NE*NNODE*sizeof(int) );

    for(int i=0; i<M*N; i++)
      A[i] = 0.0f;
    for(int i=0; i<mesh.elements.size(); i++)
    {
      Element *e = mesh.elements[i];
      X[NNODE*i+0] = e->nodes[0]->x;
      X[NNODE*i+1] = e->nodes[1]->x;
      X[NNODE*i+2] = e->nodes[2]->x;
      Y[NNODE*i+0] = e->nodes[0]->y;
      Y[NNODE*i+1] = e->nodes[1]->y;
      Y[NNODE*i+2] = e->nodes[2]->y;
      gIdx[NNODE*i+0] = e->nodes[0]->index;
      gIdx[NNODE*i+1] = e->nodes[1]->index;
      gIdx[NNODE*i+2] = e->nodes[2]->index;
    }

    float *dA = NULL;
    hipMalloc((void**)&dA, M*N*sizeof(float));
    float *dX = NULL;
    hipMalloc((void**)&dX, NE*NNODE*sizeof(float));
    float *dY = NULL;
    hipMalloc((void**)&dY, NE*NNODE*sizeof(float));
    int *dGIdx = NULL;
    hipMalloc((void**)&dGIdx, NE*NNODE*sizeof(int));

    hipMemcpy(dA, A, M*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, X, NE*NNODE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, Y, NE*NNODE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dGIdx, gIdx, NE*NNODE*sizeof(int), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    fea_kernel << <2, dim_block >> >(dA, dX, dY, dGIdx); //bugfix 1 => 2

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed, start, stop);

    printf("GPU Time: %f ms\n", elapsed);

    hipDeviceSynchronize();
    cudaStatus = hipMemcpy(A, dA, M*N*sizeof(float), hipMemcpyDeviceToHost);
    for(int i=0; i<M; i++) {
      for(int j=0; j<N; j++) {
        printf("%f ", A[i*N+j]);
      }
      printf("\n");
    }

    hipFree(dA);
    hipFree(dX);
    hipFree(dY);
    hipFree(dGIdx);

    return cudaStatus;
}

//////////////////////////////////////////////////////////////
//nvcc --std=c++11 fea_symbolic.cu -o fea_symbolic -lginac
int main()
{
    assembleWithCuda();
    hipDeviceReset();
    return 0;
}
